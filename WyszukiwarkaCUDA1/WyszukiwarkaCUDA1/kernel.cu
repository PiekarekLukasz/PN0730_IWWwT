#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <windows.h>
#include <vector>

#define BASE_CHUNK_SIZE 1024  // tego proszę nie zmieniać 
#define CHUNK_SIZE_MULTIPLIER 32 // to kręcić jak dusza zapragnie (od 1 do 2^32 czy coś)
#define CHUNK_SIZE (BASE_CHUNK_SIZE*CHUNK_SIZE_MULTIPLIER)  // tego też nie dodykać, twarda definicja

#define MAX_WORD_LENGHT 64
#define MAX_PATH_LENGHT 300

hipError_t searchFileWithCuda(std::vector<char*> analyze, const char* word, std::vector<int*>& result, unsigned int word_lenght);

__global__ void searchKernel(const char* analyze, const char* word, int* result, unsigned int word_lenght)
{
    int x = blockIdx.x;
    int i = threadIdx.x;
    if (i < CHUNK_SIZE - word_lenght)
    {
        int j;
        bool found = true;
        for (j = 0; j < word_lenght; j++)
        {
            if (analyze[(BASE_CHUNK_SIZE * x) + i + j] != word[j]) found = false;
        }
        if (found)
        {
            result[(BASE_CHUNK_SIZE * x) + i] = 1;
        }
    }
}

int readFiletoBuffer(char const* path, std::vector<char*>& chunks)
{

    FILE* f = fopen(path, "rb");
    fseek(f, 0, SEEK_END);
    long fsize = ftell(f);
    fseek(f, 0, SEEK_SET);  /* same as rewind(f); */

    while (fsize > 0)
    {
        long size_to_read = CHUNK_SIZE < fsize ? CHUNK_SIZE : fsize;
        int rollback = -MAX_WORD_LENGHT;

        char* contents = (char*)calloc(CHUNK_SIZE, sizeof(char));
        fread(contents, 1, size_to_read - 1, f);

        contents[size_to_read - 1] = '\0';
        chunks.push_back(contents);

        fseek(f, rollback, SEEK_CUR);
        fsize -= (CHUNK_SIZE + rollback - 1);
    }


    fclose(f);

    return fsize;
}

int findAllFilesInDir(char const* path, std::vector<char*>& files)
{
    char expandedpath[MAX_PATH_LENGHT];
    strcpy(expandedpath, path);
    strcat(expandedpath, "\\*.txt");

    WIN32_FIND_DATA data;
    HANDLE hFIND = FindFirstFile(expandedpath, &data);
    if (hFIND == INVALID_HANDLE_VALUE)
    {
        printf("Invalid folder path!");
        return 1;
    }
    do {
        char* fullpath = (char*)calloc(MAX_PATH_LENGHT, sizeof(char));
        strcpy(fullpath, path);
        strcat(fullpath, "\\");
        strcat(fullpath, data.cFileName);
        files.push_back(fullpath);
    } while (FindNextFile(hFIND, &data));

    return 0;
}

int main()
{
    std::vector<char*> files;

    char word[MAX_WORD_LENGHT];
    int word_lenght;

    char path[MAX_PATH_LENGHT];

    printf("Path to the folder to be searched: \n");
    scanf("%s", path);
    printf("Word to be searched for: \n");
    scanf("%s", word);

    word_lenght = strlen(word);

    findAllFilesInDir(path, files);

    for (char* file : files)
    {
        long pos_shift = 0;
        printf("FILE: %s \n", file);
        std::vector<char*> chunks;
        readFiletoBuffer(file, chunks);
        std::vector<int*> results;

        hipError_t cudaStatus = searchFileWithCuda(chunks, word, results, word_lenght);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addWithCuda failed!");
            return 1;
        }


        for (int* result : results)
        {
            for (int i = 0; i < CHUNK_SIZE; i++) if (result[i] == 1) printf("Hit on pos: %d \n", pos_shift + i);
            pos_shift += (CHUNK_SIZE - MAX_WORD_LENGHT - 1);
            free(result);
        }

        free(file);
    }


    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t searchFileWithCuda(std::vector<char*> analyze, const char* word, std::vector<int*>& result, unsigned int word_lenght)
{
    char* internal_analyze = 0;
    int* internal_result = 0;
    char* Internal_word = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&internal_result, CHUNK_SIZE * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&internal_analyze, CHUNK_SIZE * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&Internal_word, word_lenght * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(Internal_word, word, word_lenght * sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! third");
        goto Error;
    }


    for (char* contents : analyze)
    {
        cudaStatus = hipMemcpy(internal_analyze, contents, CHUNK_SIZE * sizeof(char), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed! first");
            goto Error;
        }

        cudaStatus = hipMemset(internal_result, 0, CHUNK_SIZE * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed! second");
            goto Error;
        }

        // Launch a kernel on the GPU with one thread for each element.
        searchKernel << < CHUNK_SIZE_MULTIPLIER, BASE_CHUNK_SIZE >> > (internal_analyze, Internal_word, internal_result, word_lenght);

        free(contents);

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            goto Error;
        }

        int* partial_result = (int*)calloc(CHUNK_SIZE, sizeof(int));

        // Copy output vector from GPU buffer to host memory.
        cudaStatus = hipMemcpy(partial_result, internal_result, CHUNK_SIZE * sizeof(int), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        result.push_back(partial_result);
    }


Error:
    hipFree(Internal_word);
    hipFree(internal_analyze);
    hipFree(internal_result);

    return cudaStatus;
}
